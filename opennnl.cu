#include "hip/hip_runtime.h"
#include "opennnl.h"

OpenNNL::OpenNNL(const int inputsCount, const int layersCount, const int * neuronsPerLayerCount)
{
    _inputsCount = inputsCount;
    _layersCount = layersCount;
    _weightsCount = 0;
    _neuronsCount = 0;

    _neuronsPerLayerCount = new int[_layersCount];
    _neuronsInPreviousLayers = new int[_layersCount];
    _inputsInPreviousLayers = new int[_layersCount];
    _inputsInCurrentLayer = new int[_layersCount];

    _inputs = new double[_inputsCount];

    int inputs = _inputsCount;

    for(int i=0;i<_layersCount;i++)
    {
        _neuronsInPreviousLayers[i] = _neuronsCount;
        _inputsInPreviousLayers[i] = _weightsCount;

        _inputsInCurrentLayer[i] = inputs;

        _weightsCount += neuronsPerLayerCount[i] * inputs;
        _neuronsCount += neuronsPerLayerCount[i];

        inputs = _neuronsPerLayerCount[i] = neuronsPerLayerCount[i];
    }

    _outputsCount = inputs;
    _outputs = new double[_outputsCount];

    _derivatives = new double[_neuronsCount];

    _neuronsInputsWeights = new double[_weightsCount];
    _neuronsBiases = new double[_neuronsCount];
}

OpenNNL::~OpenNNL()
{
    delete[] _neuronsPerLayerCount;
    delete[] _neuronsInPreviousLayers;
    delete[] _inputsInPreviousLayers;
    delete[] _inputsInCurrentLayer;
    delete[] _inputs;
    delete[] _outputs;
    delete[] _derivatives;
    delete[] _neuronsInputsWeights;
    delete[] _neuronsBiases;
}

void OpenNNL::printDebugInfo()
{
    printf("inputsCount=%d\n", _inputsCount);
    printf("outputsCount=%d\n", _outputsCount);
    printf("layersCount=%d\n", _layersCount);
    printf("neuronsCount=%d\n", _neuronsCount);
    printf("weightsCount=%d\n", _weightsCount);

    for(int i=0;i<_layersCount;i++)
    {
        printf("neurons in layer %d: %d\n", i, _neuronsPerLayerCount[i]);
        printf("neurons in all layers before %d: %d\n", i, _neuronsInPreviousLayers[i]);
        printf("inputs in all layers before %d: %d\n", i, _inputsInPreviousLayers[i]);
        printf("inputs of each neuron in layer %d: %d\n", i, _inputsInCurrentLayer[i]);
    }
}

inline int OpenNNL::indexByLayerAndNeuron(int layer, int neuron)
{
    return _neuronsInPreviousLayers[layer] + neuron;
}

inline int OpenNNL::indexByLayerNeuronAndInput(int layer, int neuron, int input)
{
    return _inputsInPreviousLayers[layer] + neuron*_inputsInCurrentLayer[layer] + input;
}

inline void OpenNNL::setB(int layer, int neuron, int input, double value)
{
    _Bs[indexByLayerNeuronAndInput(layer, neuron, input)] = value;
}

inline double OpenNNL::getB(int layer, int neuron, int input)
{
    return _Bs[indexByLayerNeuronAndInput(layer, neuron, input)];
}

inline void OpenNNL::setBForBias(int layer, int neuron, double value)
{
    _BsForBias[indexByLayerAndNeuron(layer, neuron)] = value;
}

inline double OpenNNL::getBForBias(int layer, int neuron)
{
    return _BsForBias[indexByLayerAndNeuron(layer, neuron)];
}

inline void OpenNNL::setH(int layer, int neuron, int input, double value)
{
    _Hs[indexByLayerNeuronAndInput(layer, neuron, input)] = value;
}

inline double OpenNNL::getH(int layer, int neuron, int input)
{
    return _Hs[indexByLayerNeuronAndInput(layer, neuron, input)];
}

inline void OpenNNL::setHForBias(int layer, int neuron, double value)
{
    _HsForBias[indexByLayerAndNeuron(layer, neuron)] = value;
}

inline double OpenNNL::getHForBias(int layer, int neuron)
{
    return _HsForBias[indexByLayerAndNeuron(layer, neuron)];
}

inline void OpenNNL::setWeight(int layer, int neuron, int input, double value)
{
    _neuronsInputsWeights[indexByLayerNeuronAndInput(layer, neuron, input)] = value;
}

inline double OpenNNL::getWeight(int layer, int neuron, int input)
{
    return _neuronsInputsWeights[indexByLayerNeuronAndInput(layer, neuron, input)];
}

inline void OpenNNL::setBias(int layer, int neuron, double value)
{
    _neuronsBiases[indexByLayerAndNeuron(layer, neuron)] = value;
}

inline double OpenNNL::getBias(int layer, int neuron)
{
    return _neuronsBiases[indexByLayerAndNeuron(layer, neuron)];
}

inline void OpenNNL::setDerivative(int layer, int neuron, double value)
{
    _derivatives[indexByLayerAndNeuron(layer, neuron)] = value;
}

inline double OpenNNL::getDerivative(int layer, int neuron)
{
    return _derivatives[indexByLayerAndNeuron(layer, neuron)];
}

inline void OpenNNL::setInput(int index, double value)
{
    _inputs[index] = value;
}

inline double OpenNNL::getOutput(int index)
{
    return _outputs[index];
}

void OpenNNL::randomizeWeights()
{
    initialize_random_generator();

    /*for(int i=0;i<_weightsCount;i++)
    {
        _neuronsInputsWeights[i] = unified_random();
    }*/

    int inputs = _inputsCount;

    for(int i=0;i<_layersCount;i++)
    {
        for(int j=0;j<inputs*_neuronsPerLayerCount[i];j++)
        {
            _neuronsInputsWeights[_inputsInPreviousLayers[i]+j] = unified_random() / sqrt(inputs);
        }

        inputs = _neuronsPerLayerCount[i];
    }

}

void OpenNNL::randomizeBiases()
{
    initialize_random_generator();

    /*for(int i=0;i<_neuronsCount;i++)
    {
        _neuronsBiases[i] = unified_random();
    }*/
    int inputs = _inputsCount;

    for(int i=0;i<_layersCount;i++)
    {
        for(int j=0;j<_neuronsPerLayerCount[i];j++)
        {
            _neuronsBiases[_neuronsInPreviousLayers[i]+j] = unified_random() / sqrt(inputs);
        }

        inputs = _neuronsPerLayerCount[i];
    }
}

void OpenNNL::randomizeWeightsAndBiases()
{
    this->randomizeWeights();
    this->randomizeBiases();
}

/*****************************************************************************/
/* Вычислить активационную функцию y(x) = 2x / (1 + abs(x)). */
/*****************************************************************************/
inline double OpenNNL::activation(double x, TActivationKind kind)
{
    return ((kind == SIG) ? (2.0 * x / (1 + fabs(x))):x);
}

/*****************************************************************************/
/* Вычислить производную активационной функции y(x) по формуле:
   dy(x)         2.0
   ----- = ---------------.
    dx     (1 + abs(x))^2
*/
/*****************************************************************************/
inline double OpenNNL::activation_derivative(double x, TActivationKind kind)
{
    double temp = 1.0 + fabs(x);
    return ((kind == SIG) ? (2.0 / (temp * temp)):1.0);
}

double * OpenNNL::_calculateWorker(double *inpt)
{
    int inputsCount;
    double * temp;
    double * inputs = new double[_inputsCount];

    memcpy(inputs, inpt, sizeof(double)*_inputsCount);

    inputsCount = _inputsCount;

    for(int i=0;i<_layersCount;i++)
    {
        temp = new double[_neuronsPerLayerCount[i]*inputsCount];
        for(int j=0;j<_neuronsPerLayerCount[i];j++)
        {
            for(int k=0;k<inputsCount;k++)
            {
            temp[j*inputsCount+k] = inputs[k] * _neuronsInputsWeights[indexByLayerNeuronAndInput(i, j, k)];
            }
        }

        delete[] inputs;

        inputs = new double[_neuronsPerLayerCount[i]];

        for(int j=0;j<_neuronsPerLayerCount[i];j++)
        {
            inputs[j] = 0;

            for(int k=0;k<inputsCount;k++)
            {
                inputs[j] += temp[j*inputsCount+k];
            }

            inputs[j]  -= _neuronsBiases[indexByLayerAndNeuron(i, j)];

            inputs[j] = activation(inputs[j]);

        }

        inputsCount = _neuronsPerLayerCount[i];
        delete[] temp;
    }

    memcpy(_outputs, inputs, sizeof(double)*inputsCount);

    delete[] inputs;

    return _outputs;
}

double * OpenNNL::calculate(double *inputs)
{
    if(inputs)
    {
        memcpy(_inputs, inputs, _inputsCount*sizeof(double));
    }

    return _calculateWorker(_inputs);
}

double * OpenNNL::calculateRef(double *inputs)
{
    if(!inputs)
        inputs = _inputs;

    return _calculateWorker(inputs);
}

void OpenNNL::calculateNeuronsOutputsAndDerivatives(double *inpt, double *outputs, double *derivatives)
{
    int inputsCount, neuronIndex = 0;
    double * temp;
    double * inputs = new double[_inputsCount];

    memcpy(inputs, inpt, sizeof(double)*_inputsCount);

    inputsCount = _inputsCount;

    for(int i=0;i<_layersCount;i++)
    {
        temp = new double[_neuronsPerLayerCount[i]*inputsCount];
        for(int j=0;j<_neuronsPerLayerCount[i];j++)
        {
            for(int k=0;k<inputsCount;k++)
            {
                temp[j*inputsCount+k] = inputs[k] * _neuronsInputsWeights[indexByLayerNeuronAndInput(i, j, k)];
            }
        }

        delete[] inputs;

        inputs = new double[_neuronsPerLayerCount[i]];

        for(int j=0;j<_neuronsPerLayerCount[i];j++)
        {
            inputs[j] = 0;

            for(int k=0;k<inputsCount;k++)
            {
                inputs[j] += temp[j*inputsCount+k];
            }

            inputs[j] -= _neuronsBiases[indexByLayerAndNeuron(i, j)];

            outputs[neuronIndex] = inputs[j] = activation(inputs[j]);
            derivatives[neuronIndex] = activation_derivative(inputs[j]);

            neuronIndex++;
        }

        inputsCount = _neuronsPerLayerCount[i];

        delete[] temp;
    }

    delete[] inputs;
}

double OpenNNL::_changeWeightsByBP(double * trainingInputs, double *trainingOutputs, double speed, double sample_weight)
{
    double error = 0, current_error;
    double * localGradients = new double[_neuronsCount];
    double * outputs = new double[_neuronsCount];
    double * derivatives = new double[_neuronsCount];

    calculateNeuronsOutputsAndDerivatives(trainingInputs, outputs, derivatives);

    for(int j=0;j<_neuronsPerLayerCount[_layersCount-1];j++) // cuda kernel
    {
        current_error = trainingOutputs[j] - outputs[indexByLayerAndNeuron(_layersCount-1, j)];
        localGradients[indexByLayerAndNeuron(_layersCount-1, j)] = current_error * sample_weight * derivatives[indexByLayerAndNeuron(_layersCount-1, j)];

        error += current_error * current_error;
    }

    if(_layersCount > 1)
    {
        for(int i=_layersCount-2;i>=0;i--)
        {
            for(int j=0;j<_neuronsPerLayerCount[i];j++) // cuda kernel
            {
                localGradients[indexByLayerAndNeuron(i, j)] = 0;

                for(int k=0;k<_neuronsPerLayerCount[i+1];k++)
                {
                    localGradients[indexByLayerAndNeuron(i, j)] += _neuronsInputsWeights[indexByLayerNeuronAndInput(i+1, k, j)]
                                                                    * localGradients[indexByLayerAndNeuron(i+1, k)];
                }

                localGradients[indexByLayerAndNeuron(i, j)] *= derivatives[indexByLayerAndNeuron(i, j)];
            }
        }
    }

    for(int j=0;j<_neuronsPerLayerCount[0];j++) // this and next cicle for cuda kernel (j*k threads)
    {
        for(int k=0;k<_inputsCount;k++)
        {
            _neuronsInputsWeights[indexByLayerNeuronAndInput(0, j, k)] += speed * localGradients[indexByLayerAndNeuron(0, j)] * trainingInputs[k];
        }

        _neuronsBiases[indexByLayerAndNeuron(0, j)] -= speed * localGradients[indexByLayerAndNeuron(0, j)];
    }

    for(int i=1;i<_layersCount;i++) // try to parallelize all three cicles in one kernel. If it's impossible, only two inner
    {
        for(int j=0;j<_neuronsPerLayerCount[i];j++)
        {
            for(int k=0;k<_neuronsPerLayerCount[i-1];k++)
            {
                _neuronsInputsWeights[indexByLayerNeuronAndInput(i, j, k)] += speed * localGradients[indexByLayerAndNeuron(i, j)] * outputs[indexByLayerAndNeuron(i-1, k)];
            }

            _neuronsBiases[indexByLayerAndNeuron(i, j)] -= speed * localGradients[indexByLayerAndNeuron(i, j)];
        }
    }

    delete[] localGradients;
    delete[] outputs;
    delete[] derivatives;

    error /= 2;
    return error;
}

double OpenNNL::_changeWeightsByIDBD(double * trainingInputs, double *trainingOutputs, double speed, double sample_weight)
{
    double error = 0, current_error;
    double cur_rate, delta, deltaB, deltaH;
    double * localGradients = new double[_neuronsCount];
    double * outputs = new double[_neuronsCount];
    double * derivatives = new double[_neuronsCount];

    calculateNeuronsOutputsAndDerivatives(trainingInputs, outputs, derivatives);

    for(int j=0;j<_neuronsPerLayerCount[_layersCount-1];j++)
    {
        current_error = trainingOutputs[j] - outputs[indexByLayerAndNeuron(_layersCount-1, j)];
        localGradients[indexByLayerAndNeuron(_layersCount-1, j)] = current_error * sample_weight * derivatives[indexByLayerAndNeuron(_layersCount-1, j)];

        error += current_error * current_error;
    }

    if(_layersCount > 1)
    {
        for(int i=_layersCount-2;i>=0;i--)
        {
            for(int j=0;j<_neuronsPerLayerCount[i];j++)
            {
                localGradients[indexByLayerAndNeuron(i, j)] = 0;

                for(int k=0;k<_neuronsPerLayerCount[i+1];k++)
                {
                    localGradients[indexByLayerAndNeuron(i, j)] += _neuronsInputsWeights[indexByLayerNeuronAndInput(i+1, k, j)]
                                                                    * localGradients[indexByLayerAndNeuron(i+1, k)];
                }

                localGradients[indexByLayerAndNeuron(i, j)] *= derivatives[indexByLayerAndNeuron(i, j)];
            }
        }
    }

    for(int j=0;j<_neuronsPerLayerCount[0];j++)
    {
        for(int k=0;k<_inputsCount;k++)
        {
            deltaB = speed * localGradients[indexByLayerAndNeuron(0, j)] * trainingInputs[k] * getH(0, j, k);

            if (deltaB > 2.0)
            {
                deltaB = 2.0;
            }
            else
            {
                if (deltaB < -2.0)
                {
                    deltaB = -2.0;
                }
            }

            setB(0, j, k, getB(0, j, k) + deltaB);
            cur_rate = exp(getB(0, j, k));
            delta = cur_rate * localGradients[indexByLayerAndNeuron(0, j)] * trainingInputs[k];

            _neuronsInputsWeights[indexByLayerNeuronAndInput(0, j, k)] += delta;

            deltaH = 1 - cur_rate * trainingInputs[k] * trainingInputs[k];
            if(deltaH <= 0)
                setH(0, j, k, delta);
            else
                setH(0, j, k, getH(0, j, k) * deltaH + delta);
        }

        deltaB = speed * localGradients[indexByLayerAndNeuron(0, j)] * getHForBias(0, j);

        if (deltaB > 2.0)
        {
            deltaB = 2.0;
        }
        else
        {
            if (deltaB < -2.0)
            {
                deltaB = -2.0;
            }
        }

        setBForBias(0, j, getBForBias(0, j) - deltaB);
        cur_rate = exp(getBForBias(0, j));
        delta = cur_rate * localGradients[indexByLayerAndNeuron(0, j)];

        _neuronsBiases[indexByLayerAndNeuron(0, j)] -= delta;

        deltaH = 1 - cur_rate;
        if(deltaH <= 0)
            setHForBias(0, j, -delta);
        else
            setHForBias(0, j, getHForBias(0, j) * deltaH - delta);
    }

    for(int i=1;i<_layersCount;i++)
    {
        for(int j=0;j<_neuronsPerLayerCount[i];j++)
        {
            for(int k=0;k<_neuronsPerLayerCount[i-1];k++)
            {
                deltaB = speed * localGradients[indexByLayerAndNeuron(i, j)] * outputs[indexByLayerAndNeuron(i-1, k)] * getH(i, j, k);

                if (deltaB > 2.0)
                {
                    deltaB = 2.0;
                }
                else
                {
                    if (deltaB < -2.0)
                    {
                        deltaB = -2.0;
                    }
                }

                setB(i, j, k, getB(i, j, k) + deltaB);
                cur_rate = exp(getB(i, j, k));
                delta = cur_rate * localGradients[indexByLayerAndNeuron(i, j)] * outputs[indexByLayerAndNeuron(i-1, k)];

                _neuronsInputsWeights[indexByLayerNeuronAndInput(i, j, k)] += delta;

                deltaH = 1 - cur_rate * outputs[indexByLayerAndNeuron(i-1, k)] * outputs[indexByLayerAndNeuron(i-1, k)];
                if(deltaH <= 0)
                    setH(i, j, k, delta);
                else
                    setH(i, j, k, getH(i, j, k) * deltaH + delta);
            }

            deltaB = speed * localGradients[indexByLayerAndNeuron(i, j)] * getHForBias(i, j);

            if (deltaB > 2.0)
            {
                deltaB = 2.0;
            }
            else
            {
                if (deltaB < -2.0)
                {
                    deltaB = -2.0;
                }
            }

            setBForBias(i, j, getBForBias(i, j) - deltaB);
            cur_rate = exp(getBForBias(i, j));
            delta = cur_rate * localGradients[indexByLayerAndNeuron(i, j)];

            _neuronsBiases[indexByLayerAndNeuron(i, j)] -= delta;

            deltaH = 1 - cur_rate;
            if(deltaH <= 0)
                setHForBias(i, j, -delta);
            else
                setHForBias(i, j, getHForBias(i, j) * deltaH - delta);
        }
    }

    delete[] localGradients;
    delete[] outputs;
    delete[] derivatives;

    error /= 2;
    return error;
}

/*double OpenNNL::_changeWeightsByIDBD(double * trainingInputs, double *trainingOutputs, double speed, double sample_weight)
{
    int i, j, k, nInputsCount;
    double cur_output, cur_input, cur_error;
    double delta_bias, delta_weight;
    double cur_rate, dB, newH;

    double * localGradients = new double[_neuronsCount];
    double * outputs = new double[_neuronsCount];
    double * derivatives = new double[_neuronsCount];

    calculateNeuronsOutputsAndDerivatives(trainingInputs, outputs, derivatives);

    if(_layersCount > 1)
    {
        i = _layersCount-1;
        nInputsCount = _inputsInCurrentLayer[i];

        for (j = 0; j < _neuronsPerLayerCount[i]; j++)
        {
            cur_error = (trainingOutputs[j] - outputs[indexByLayerAndNeuron(i, j)]) * sample_weight;

            localGradients[indexByLayerAndNeuron(i, j)] = cur_error * derivatives[indexByLayerAndNeuron(i, j)];

            dB = speed * localGradients[indexByLayerAndNeuron(i, j)] * getHForBias(i, j);

            if (dB > 2.0)
            {
                dB = 2.0;
            }
            else
            {
                if (dB < -2.0)
                {
                    dB = -2.0;
                }
            }
            setBForBias(i, j, getBForBias(i, j) + dB);
            cur_rate = exp(getBForBias(i, j));

            delta_bias = cur_rate * localGradients[indexByLayerAndNeuron(i, j)];
            setBias(i, j, getBias(i, j) + delta_bias);

            newH = 1.0 - cur_rate;
            if (newH <= 0.0)
            {
                newH = delta_bias;
            }
            else
            {
                newH = getHForBias(i ,j) * newH + delta_bias;
            }
            setHForBias(i, j, newH);
        }

        // Цикл по всем скрытым слоям от последнего до первого
        for (i = _layersCount-2; i >= 0; i--)
        {
            nInputsCount = _inputsInCurrentLayer[i];

            for (j = 0; j < _neuronsPerLayerCount[i]; j++)
            {
                cur_output = outputs[indexByLayerAndNeuron(i, j)];
                cur_error = 0.0;
                for (k = 0; k < _neuronsPerLayerCount[i+1]; k++)
                {
                    cur_error += getWeight(i+1,k,j) * localGradients[indexByLayerAndNeuron(i, k)];

                    dB = speed * localGradients[indexByLayerAndNeuron(i, k)] * getH(i+1,k,j) * cur_output;
                    if (dB > 2.0)
                    {
                        dB = 2.0;
                    }
                    else
                    {
                        if (dB < -2.0)
                        {
                            dB = -2.0;
                        }
                    }
                    setB(i+1,k,j, getB(i+1,k,j) + dB);

                    cur_rate = exp(getB(i+1,k,j));
                    //cur_rate = m_rate;

                    delta_weight = cur_rate*cur_output*localGradients[indexByLayerAndNeuron(i, k)];
                    setWeight(i+1,k,j, getWeight(i+1,k,j) + delta_weight);

                    newH = 1.0 - cur_rate * cur_output * cur_output;
                    if (newH <= 0.0)
                    {
                        newH = delta_weight;
                    }
                    else
                    {
                        newH = getH(i+1,k,j) * newH + delta_weight;
                    }
                    setH(i+1,k,j, newH);
                }

                // на основе ошибки вычисляем локальный градиент
                localGradients[indexByLayerAndNeuron(i, j)] = cur_error * derivatives[indexByLayerAndNeuron(i, j)];

                dB = speed * localGradients[indexByLayerAndNeuron(i, j)] * getHForBias(i, j);
                if (dB > 2.0)
                {
                    dB = 2.0;
                }
                else
                {
                    if (dB < -2.0)
                    {
                        dB = -2.0;
                    }
                }
                setBForBias(i, j, getBForBias(i,j)+dB);
                cur_rate = exp(getBForBias(i, j));

                // корректируем смещение нейрона
                delta_bias = cur_rate * localGradients[indexByLayerAndNeuron(i, j)];

                setBias(i, j, getBias(i,j) + delta_bias);

                // вычисляем новое значение параметра H для смещения
                newH = 1.0 - cur_rate;
                if (newH <= 0.0)
                {
                    newH = delta_bias;
                }
                else
                {
                    newH = getHForBias(i, j) * newH + delta_bias;
                }
                setHForBias(i, j, newH);
            }
        }

        for (j = 0; j < _neuronsPerLayerCount[0]; j++)
        {
            for (k = 0; k < nInputsCount; k++)
            {
                dB = speed * localGradients[indexByLayerAndNeuron(0, j)] * getH(0,j,k) * trainingInputs[k];
                if (dB > 2.0)
                {
                    dB = 2.0;
                }
                else
                {
                    if (dB < -2.0)
                    {
                        dB = -2.0;
                    }
                }
                setB(0,j,k, getB(0,j,k) + dB);

                cur_rate = exp(getB(0,j,k));
                //cur_rate = m_rate;

                cur_input = trainingInputs[k];

                delta_weight = cur_rate * cur_input * localGradients[indexByLayerAndNeuron(0, j)];
                setWeight(0, j, k, getWeight(0, j, k) + delta_weight);

                newH = 1.0 - cur_rate * cur_input * cur_input;
                if (newH <= 0.0)
                {
                    newH = delta_weight;
                }
                else
                {
                    newH = getH(0,j,k) * newH + delta_weight;
                }
                setH(0,j,k, newH);
            }
        }
    }
    else
    {
        nInputsCount = _inputsInCurrentLayer[0];
        // Для каждого нейрона слоя (цикл по j)
        for (j = 0; j < _neuronsPerLayerCount[0]; j++)
        {
            cur_error = (trainingOutputs[j] - outputs[indexByLayerAndNeuron(0, j)]) * sample_weight;

            // вычисляем локальный градиент
            localGradients[indexByLayerAndNeuron(0, j)] = cur_error * derivatives[indexByLayerAndNeuron(0, j)];

            dB = speed * localGradients[indexByLayerAndNeuron(0, j)] * getHForBias(0,j);
            if (dB > 2.0)
            {
                dB = 2.0;
            }
            else
            {
                if (dB < -2.0)
                {
                    dB = -2.0;
                }
            }
            setBForBias(0,j, getBForBias(0,j) + dB);
            cur_rate = exp(getBForBias(0,j));

            // корректируем смещение нейрона
            delta_bias = cur_rate * localGradients[indexByLayerAndNeuron(0, j)];
            setBias(0, j, getBias(0, j) + delta_bias);

            // вычисляем новое значение параметра H для смещения
            newH = 1.0 - cur_rate;
            if (newH <= 0.0)
            {
                newH = delta_bias;
            }
            else
            {
                newH = getHForBias(0, j) * newH + delta_bias;
            }
            setHForBias(0, j, newH);

            // Для всех входов j-го нейрона (цикл по k)
            for (k = 0; k < nInputsCount; k++)
            {
                dB = speed * localGradients[indexByLayerAndNeuron(0, j)] * getH(0,j,k) * trainingInputs[k];
                if (dB > 2.0)
                {
                    dB = 2.0;
                }
                else
                {
                    if (dB < -2.0)
                    {
                        dB = -2.0;
                    }
                }
                setB(0, j, k, getB(0, j, k) + dB);

                cur_rate = exp(getB(0, j, k));

                cur_input = trainingInputs[k];

                delta_weight = cur_rate * cur_input * localGradients[indexByLayerAndNeuron(0, j)];
                setWeight(0,j,k, getWeight(0,j,k) + delta_weight);

                newH = 1.0 - cur_rate * cur_input * cur_input;
                if (newH <= 0.0)
                {
                    newH = delta_weight;
                }
                else
                {
                    newH = getH(0,j,k) * newH + delta_weight;
                }
                setH(0,j,k, newH);
            }
        }
    }

    delete[] localGradients;
    delete[] outputs;
    delete[] derivatives;
}*/

bool OpenNNL::_doEpochBP(int samplesCount, double * trainingInputs, double * trainingOutputs, int numEpoch, double speed, double minError)
{
    double error = 0;
    double * currentSampleInputs = new double[_inputsCount];
    double * currentSampleOutputs = new double[_outputsCount];

    for(int sample=0;sample<samplesCount;sample++)
    {
        cout << "Epoch: " << numEpoch << ", Sample: " << sample << endl;
        memcpy(currentSampleInputs, trainingInputs+sample*_inputsCount, _inputsCount*sizeof(double));
        memcpy(currentSampleOutputs, trainingOutputs+sample*_outputsCount, _outputsCount*sizeof(double));

        error = _changeWeightsByBP(currentSampleInputs, currentSampleOutputs, speed, 1);
    }

    delete[] currentSampleInputs;
    delete[] currentSampleOutputs;

    return (error <= minError);
}

bool OpenNNL::_doEpochIDBD(int samplesCount, double * trainingInputs, double * trainingOutputs, int numEpoch, double speed, double minError)
{
    double error = 0;
    double * currentSampleInputs = new double[_inputsCount];
    double * currentSampleOutputs = new double[_outputsCount];

    for(int sample=0;sample<samplesCount;sample++)
    {
        cout << "Sample: " << sample << endl;
        memcpy(currentSampleInputs, trainingInputs+sample*_inputsCount, _inputsCount*sizeof(double));
        memcpy(currentSampleOutputs, trainingOutputs+sample*_outputsCount, _outputsCount*sizeof(double));

        error = _changeWeightsByIDBD(currentSampleInputs, currentSampleOutputs, speed, 1);
    }

    delete[] currentSampleInputs;
    delete[] currentSampleOutputs;

    return (error <= minError);
}

void OpenNNL::_trainingBP(int samplesCount, double * trainingInputs, double * trainingOutputs, int maxEpochsCount, double speed, double error)
{
    for(int i=0;i<maxEpochsCount;i++)
    {
        if(_doEpochBP(samplesCount, trainingInputs, trainingOutputs, i, speed, error))
        {
            break;
        }
    }
}


void OpenNNL::_trainingIDBD(int samplesCount, double * trainingInputs, double * trainingOutputs, int maxEpochsCount, double speed, double error)
{
    for(int i=0;i<maxEpochsCount;i++)
    {
        if(_doEpochIDBD(samplesCount, trainingInputs, trainingOutputs, i, speed, error))
        {
            break;
        }
    }
}

void OpenNNL::trainingBP(int samplesCount, double * trainingInputs, double *trainingOutputs, int maxEpochsCount, double speed, double error)
{
    _trainingBP(samplesCount, trainingInputs, trainingOutputs, maxEpochsCount, speed, error);
}

void OpenNNL::trainingIDBD(int samplesCount, double * trainingInputs, double *trainingOutputs, int maxEpochsCount, double speed, double error)
{
    _Bs = new double[_weightsCount];
    _Hs = new double[_weightsCount];

    _BsForBias = new double[_neuronsCount];
    _HsForBias = new double[_neuronsCount];

    resetHsAndHsForBias();
    randomizeBsAndBsForBias();

    _trainingIDBD(samplesCount, trainingInputs, trainingOutputs, maxEpochsCount, speed, error);

    delete[] _Bs;
    delete[] _Hs;
    delete[] _BsForBias;
    delete[] _HsForBias;
}

void OpenNNL::getOutputs(double * out)
{
    memcpy(out, _outputs, sizeof(double)*_outputsCount);
}

void OpenNNL::resetHs()
{
    for(int i=0;i<_weightsCount;i++)
        _Hs[i] = 0;
}

void OpenNNL::resetHsForBias()
{
    for(int i=0;i<_neuronsCount;i++)
        _HsForBias[i] = 0;
}

void OpenNNL::resetHsAndHsForBias()
{
    resetHs();
    resetHsForBias();
}

void OpenNNL::randomizeBs()
{
    initialize_random_generator();
    for(int i=0;i<_weightsCount;i++)
        _Bs[i] = unified_random();
}

void OpenNNL::randomizeBsForBias()
{
    initialize_random_generator();
    for(int i=0;i<_neuronsCount;i++)
        _BsForBias[i] = unified_random();
}

void OpenNNL::randomizeBsAndBsForBias()
{
    randomizeBs();
    randomizeBsForBias();
}
